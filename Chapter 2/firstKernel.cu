#include <stdio.h>
#include <stdlib.h>



void vecAdd(float* A_h, float* B_h, float* C_h, int n){
    int size = n * sizeof(float);

    float* A_d, B_d, C_d;
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    //kernel here
    //...
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(){
    // vecAdd(A, B, C, N);
    int n = 100;
    float* A_d = (float*)malloc(n * sizeof(float));
    A_d[0] = 100.0;
    printf("%p\n", &A_d);
    printf("%p\n", *A_d);
    printf("%p\n", (void**)&A_d);
    printf("%p\n", some_addr);
    hipMalloc((void**)&A_d, n * sizeof(float));

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);

    hipFree(A_d);
}
